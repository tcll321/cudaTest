#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

typedef unsigned int    uint32;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__device__ uint32 Cloamp_10bit(float data)
{
	return ((uint32)data >> 2);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

typedef unsigned int    uint32;
typedef int             int32;
typedef unsigned char   uint8;
typedef struct uint24
{
	uint8 r;
	uint8 g;
	uint8 b;
};

__global__ void cutPicture(uint32* dstImage, uint32* srcImage, int srcWidth, int srcHeight, int cx, int cy, int width)
{
	int32 x, y;
	int srcPitch = srcWidth * 3;
	int dstPitch = width * 3;
	uint24 *srcImageU8 = (uint24 *)srcImage;
	uint24 *dstImageU8 = (uint24 *)dstImage;

	x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	y = blockIdx.y *  blockDim.y + threadIdx.y;

	if ((x) >= width)
		return; //x = width - 1;

	if ((y) >= width)
		return; // y = height - 1;

	dstImageU8[y * width + x].r = srcImageU8[(y+cy) * srcWidth + x + cx].r;
	dstImageU8[y * width + x].g = srcImageU8[(y+cy) * srcWidth + x + cx].g;
	dstImageU8[y * width + x].b = srcImageU8[(y+cy) * srcWidth + x + cx].b;
	dstImageU8[y * width + x + 1].r = srcImageU8[(y + cy) * srcWidth + x + cx + 1].r;
	dstImageU8[y * width + x + 1].g = srcImageU8[(y + cy) * srcWidth + x + cx + 1].g;
	dstImageU8[y * width + x + 1].b = srcImageU8[(y + cy) * srcWidth + x + cx + 1].b;
}

int main()
{
	hipError_t cudaStatus;
	int size = 0;
	int x = 900;
	int y = 400;
	int width = 400;
	int dstImageSize = width*width * 4;
	FILE* pf = fopen("d:\\image.rgba", "rb");
	unsigned char* pData = NULL;
	if (pf)
	{
		fseek(pf, 0L, SEEK_END);
		size = ftell(pf);
		fseek(pf, 0L, SEEK_SET);
		pData = new unsigned char[size];
		fread(pData, size, 1, pf);
		fclose(pf);
	}
	if (pData)
	{
		unsigned char* pSrcImage = NULL;
		unsigned char* pDstImage = NULL;
		unsigned char* pHImage = new unsigned char[dstImageSize];
		cudaStatus = hipMalloc((void**)&pSrcImage, size);
		cudaStatus = hipMalloc((void**)&pDstImage, dstImageSize);
		cudaStatus = hipMemcpy(pSrcImage, pData, size, hipMemcpyHostToDevice);
		dim3 block(32, 16, 1);
		dim3 grid((width + (2 * block.x - 1)) / (2 * block.x), (width + (block.y - 1)) / block.y, 1);
		cutPicture << <grid, block, 0 >> > ((uint32*)pDstImage, (uint32*)pSrcImage, 1920, 1080, x, y, width);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		}
		cudaStatus = hipMemcpy(pHImage, pDstImage, dstImageSize, hipMemcpyDeviceToHost);
		FILE* pf2 = fopen("d:\\cutimage.rgb", "wb");
		if (pf2)
		{
			fwrite(pHImage, dstImageSize, 1, pf2);
			fclose(pf2);
		}
		delete[] pHImage;
		hipFree(pSrcImage);
		hipFree(pDstImage);
		delete[] pData;
	}
	return 0;
}